
#include <hip/hip_runtime.h>
/*
 * File: \Dev-EvalFlow-v2.cu
 * Created Date: Tuesday, March 21st 2023, 8:04:17 am
 * Author: Christian Perwass (CR/AEC5)
 * <LICENSE id="Apache-2.0">
 *
 *   Image-Render Standard Actions module
 *   Copyright 2022 Robert Bosch GmbH and its subsidiaries
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 *
 * </LICENSE>
 */

#define ZERO 1e-9

template <typename T>
__device__ void inline swap(T &a, T &b)
{
    T c(a);
    a = b;
    b = c;
}

template <typename T>
__device__ void inline swap_if_valid(T &a, T &b, bool bValid)
{
    const T c(a);
    const T tT = T(bValid);
    const T tF = T(!!bValid);

    a = b * tT + a * tF;
    b = c * tT + b * tF;
}

///////////////////////////////////////////////////////////////////////////////////
// float

__device__ bool inline is_zero(const float &fX)
{
    return abs(fX) <= ZERO;
}

__device__ bool inline is_negative(const float &fX)
{
    return fX < -ZERO;
}

__device__ bool inline is_positive(const float &fX)
{
    return fX > ZERO;
}

///////////////////////////////////////////////////////////////////////////////////
// float2

__device__ float2 inline make_float2(const float *pData, const int iIdx)
{
    float2 vX = {pData[iIdx], pData[iIdx + 1]};
    return vX;
}

__device__ float2 inline operator-(const float2 &vA, const float2 &vB)
{
    float2 vX = {vA.x - vB.x, vA.y - vB.y};
    return vX;
}

__device__ float2 inline operator+(const float2 &vA, const float2 &vB)
{
    float2 vX = {vA.x + vB.x, vA.y + vB.y};
    return vX;
}

__device__ float2 inline operator*(const float &fS, const float2 &vA)
{
    float2 vX = {fS * vA.x, fS * vA.y};
    return vX;
}

__device__ float2 inline operator*(const float2 &vA, const float &fS)
{
    float2 vX = {fS * vA.x, fS * vA.y};
    return vX;
}

__device__ float inline dot(const float2 &vX, const float2 &vY)
{
    return vX.x * vY.x + vX.y * vY.y;
}

__device__ float inline sumsq(const float2 &vX)
{
    return dot(vX, vX);
}

__device__ float inline length(const float2 &vX)
{
    return sqrt(sumsq(vX));
}

///////////////////////////////////////////////////////////////////////////////////
// float3

__device__ float3 inline make_float3(const float *pData)
{
    float3 vX = {pData[0], pData[1], pData[2]};
    return vX;
}

__device__ float3 inline make_float3(const float *pData, const int iIdx)
{
    float3 vX = {pData[iIdx], pData[iIdx + 1], pData[iIdx + 2]};
    return vX;
}

__device__ void inline assign_float3(float *pData, const int iIdx, const float3 &vX)
{
    pData[iIdx + 0] = vX.x;
    pData[iIdx + 1] = vX.y;
    pData[iIdx + 2] = vX.z;
}

__device__ float inline dot(const float3 &vX, const float3 &vY)
{
    return vX.x * vY.x + vX.y * vY.y + vX.z * vY.z;
}

__device__ float inline sumsq(const float3 &vX)
{
    return dot(vX, vX);
}

__device__ float inline length(const float3 &vX)
{
    return sqrt(sumsq(vX));
}

__device__ float3 inline operator-(const float3 &vA, const float3 &vB)
{
    float3 vX = {vA.x - vB.x, vA.y - vB.y, vA.z - vB.z};
    return vX;
}

__device__ float3 inline operator+(const float3 &vA, const float3 &vB)
{
    float3 vX = {vA.x + vB.x, vA.y + vB.y, vA.z + vB.z};
    return vX;
}

__device__ float3 inline operator*(const float &fS, const float3 &vA)
{
    float3 vX = {fS * vA.x, fS * vA.y, fS * vA.z};
    return vX;
}

__device__ float3 inline operator*(const float3 &vA, const float &fS)
{
    float3 vX = {fS * vA.x, fS * vA.y, fS * vA.z};
    return vX;
}

__device__ float3 inline operator/(const float3 &vA, const float &fS)
{
    float3 vX = {vA.x / fS, vA.y / fS, vA.z / fS};
    return vX;
}

__device__ bool inline is_zero(const float3 &vX)
{
    return is_zero(sumsq(vX));
}

////////////////////////////////////////////////////////////////////////////////////////////////
// This is an inefficient (from a CUDA point of view) evaluation of optical flow from object ids and
// rendered local coordinates of the objects. However, it is still much faster than
// programming this directly in python.
template <int t_iStartX, int t_iStartY, int t_iRangeX, int t_iRangeY,
          int t_iSizeX, int t_iSizeY,
          int t_iFilterRadiusX, int t_iFilterRadiusY,
          int t_iRowStrideImage, int t_iRowStrideFlow,
          int t_iChanCntImage, int t_iChanCntFlow>
__global__ void EvalMotionBlur(const float *aImage, const float *aImage2, const float *aFlow, const float fFlowFactor, float *aResult)
{
    const float fNaN = nanf("");

    const int iTrgX = blockDim.x * blockIdx.x + threadIdx.x;
    const int iX = min(iTrgX, t_iRangeX - 1);

    const int iTrgY = blockIdx.y;
    const int iY = min(iTrgY, t_iRangeY - 1);

    const int iTrgIdxX1 = iX + t_iStartX;
    const int iIdxX1 = min(max(iTrgIdxX1, 0), t_iSizeX - 1);

    const int iTrgIdxY1 = iY + t_iStartY;
    const int iIdxY1 = min(max(iTrgIdxY1, 0), t_iSizeY - 1);

    // const int iFlowPixelIndex = iIdxY1 * t_iRowStrideFlow + iIdxX1 * t_iChanCntFlow;
    // const float2 vFlow = fFlowFactor * make_float2(aFlow, iFlowPixelIndex);

    // const bool bValidFlow = aFlow[iFlowPixelIndex + 3] > 0.5;
    const bool bValidSrcIdx = (iTrgIdxX1 == iIdxX1 && iTrgIdxY1 == iIdxY1 && iTrgX == iX && iTrgY == iY);

    const int iImagePixelIndex = iIdxY1 * t_iRowStrideImage + iIdxX1 * t_iChanCntImage;
    float3 vImgValCtr = make_float3(0.0, 0.0, 0.0);  // make_float3(aImage, iImagePixelIndex);
    float3 vImgValCtr2 = make_float3(0.0, 0.0, 0.0); // make_float3(aImage2, iImagePixelIndex);

    // float fMinValue = 1e9;
    // int iMinPosIdx2 = -1;
    // int iMinObjIdx2 = -1;

    const int iTimeBuckets = 10;
    int piTimeFlag[iTimeBuckets];
    for (int i = 0; i < iTimeBuckets; i++)
    {
        piTimeFlag[i] = 0;
    }

    const float fSigma = 2.0;
    const float fNormFac = -0.5 / (fSigma * fSigma);

    float3 vImgValResult = make_float3(0.0, 0.0, 0.0);
    float fImgValWeightSum = 0.0;
    float fImgCtrWeightSum = 0.0;
    float fTimeSum = 0.0;
    float fStartWeightSum = 0.0;
    float fEndWeightSum = 0.0;

    for (int iOffY = -t_iFilterRadiusY; iOffY <= t_iFilterRadiusY; iOffY++)
    {
        const int iIterIdxY2 = iIdxY1 + iOffY;
        const int iIdxY2 = min(max(iIterIdxY2, 0), t_iSizeY - 1);
        bool bValidY = (bValidSrcIdx && iIterIdxY2 == iIdxY2);

        for (int iOffX = -t_iFilterRadiusX; iOffX <= t_iFilterRadiusX; iOffX++)
        {
            const int iIterIdxX2 = iIdxX1 + iOffX;
            const int iIdxX2 = min(max(iIterIdxX2, 0), t_iSizeX - 1);

            const int iFlowPixelIndex = iIdxY2 * t_iRowStrideFlow + iIdxX2 * t_iChanCntFlow;
            float2 vFlow = make_float2(aFlow, iFlowPixelIndex);

            const bool bValidFlow = aFlow[iFlowPixelIndex + 3] > 0.5;

            // if (!bValidFlow)
            // {
            //     vFlow = make_float2(0.0, 0.0);
            // }

            bool bValid = (bValidY && iIterIdxX2 == iIdxX2);

            if (bValid)
            {
                const int iImagePixelIndex = iIdxY2 * t_iRowStrideImage + iIdxX2 * t_iChanCntImage;
                const float3 vImgValAtRadius = make_float3(aImage, iImagePixelIndex);
                const float3 vImgValAtRadius2 = make_float3(aImage2, iImagePixelIndex);

                float2 vRelPos = make_float2(float(iOffX), float(iOffY));

                const float fW = exp(fNormFac * sumsq(vRelPos));
                fImgCtrWeightSum += fW;

                vImgValCtr = vImgValCtr + fW * vImgValAtRadius;
                vImgValCtr2 = vImgValCtr2 + fW * vImgValAtRadius2;

                if (bValidFlow)
                {

                    const float fFlowLen2 = sumsq(vFlow);
                    float fAlpha;
                    float2 vNearPos;
                    if (is_zero(fFlowLen2))
                    {
                        fAlpha = 0.0;
                        vNearPos = vRelPos;
                    }
                    else
                    {
                        fAlpha = -dot(vFlow, vRelPos) / fFlowLen2;
                        vNearPos = vRelPos + fAlpha * vFlow;
                    }

                    if (fAlpha >= 0.0 && fAlpha <= fFlowFactor)
                    {
                        const float fTimeDelta = 1.0 / (1.0 + sqrt(fFlowLen2));
                        fTimeSum += fTimeDelta;
                        const float fInfluence = exp(fNormFac * sumsq(vNearPos));
                        const float fWeight = fInfluence * fTimeDelta;
                        fImgValWeightSum += fWeight;
                        vImgValResult = vImgValResult + fWeight * vImgValAtRadius;

                        if (fInfluence > 0.6)
                        {
                            int iIdx = min(iTimeBuckets - 1, int(floor(fAlpha * float(iTimeBuckets) + 0.5)));
                            piTimeFlag[iIdx] = 1;
                        }

                        if (fAlpha < 0.1)
                        {
                            fStartWeightSum += fWeight;
                        }
                        else if (fAlpha > 0.9)
                        {
                            fEndWeightSum += fWeight;
                        }
                    }
                }
            }
        }
    }

    if (bValidSrcIdx)
    {
        int iSum = 0;
        int iFirstSlot = iTimeBuckets;
        int iLastSlot = 0;

        for (int i = 0; i < iTimeBuckets; i++)
        {
            int iValue = piTimeFlag[i];
            if (iValue > 0)
            {
                iFirstSlot = min(i, iFirstSlot);
                iLastSlot = max(i, iLastSlot);
            }
            iSum += iValue;
        }

        // vImgValResult = make_float3(float(iFirstSlot), float(iLastSlot), 0.0);
        // fImgValWeightSum = 1.0;

        vImgValCtr = vImgValCtr / fImgCtrWeightSum;
        vImgValCtr2 = vImgValCtr2 / fImgCtrWeightSum;

        if (iLastSlot < iFirstSlot)
        {
            vImgValResult = vImgValResult + 0.5 * (vImgValCtr + vImgValCtr2);
            fImgValWeightSum += 1.0;
            // vImgValResult = make_float3(0.0, 0.0, 0.0);
        }
        else
        {
            float fWeightStart = float(iFirstSlot) / float(iTimeBuckets);
            float fWeightEnd = float(iTimeBuckets - iLastSlot - 1) / float(iTimeBuckets);
            vImgValResult = vImgValResult + 0.5 * (fWeightStart * vImgValCtr + fWeightEnd * vImgValCtr2);
            fImgValWeightSum += 0.5 * (fWeightStart + fWeightEnd);
        }

        // float fEndWeight = 1.0 - float(iSum) / float(iTimeBuckets);
        // vImgValResult = vImgValResult + fEndWeight * vImgValCtr2;

        vImgValResult = vImgValResult / fImgValWeightSum;

        // if (fStartWeightSum < 1.0)
        // {
        //     float fW = max(0.0, 1.0 - fStartWeightSum);
        //     vImgValResult = vImgValResult + fW * vImgValCtr;
        // }

        // if (fEndWeightSum < 1.0)
        // {
        //     float fW = max(0.0, 1.0 - fEndWeightSum);
        //     vImgValResult = vImgValResult + fW * vImgValCtr2;
        // }

        assign_float3(aResult, iImagePixelIndex, vImgValResult);

    } // if (bValid)
    // else
    // {

    // }
}
