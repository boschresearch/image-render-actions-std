
#include <hip/hip_runtime.h>
/*
 * File: \Dev-EvalFlow-v2.cu
 * Created Date: Tuesday, March 21st 2023, 8:04:17 am
 * Author: Christian Perwass (CR/AEC5)
 * <LICENSE id="Apache-2.0">
 *
 *   Image-Render Standard Actions module
 *   Copyright 2022 Robert Bosch GmbH and its subsidiaries
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 *
 * </LICENSE>
 */

#define ZERO 1e-12

template <typename T>
__device__ void inline swap(T &a, T &b)
{
    T c(a);
    a = b;
    b = c;
}

///////////////////////////////////////////////////////////////////////////////////
// float2

__device__ float2 inline make_float2(const float *pData, const int iIdx)
{
    float2 vX = {pData[iIdx], pData[iIdx + 1]};
    return vX;
}

__device__ float2 inline operator-(const float2 &vA, const float2 &vB)
{
    float2 vX = {vA.x - vB.x, vA.y - vB.y};
    return vX;
}

__device__ float2 inline operator+(const float2 &vA, const float2 &vB)
{
    float2 vX = {vA.x + vB.x, vA.y + vB.y};
    return vX;
}

__device__ float2 inline operator*(const float &fS, const float2 &vA)
{
    float2 vX = {fS * vA.x, fS * vA.y};
    return vX;
}

__device__ float2 inline operator*(const float2 &vA, const float &fS)
{
    float2 vX = {fS * vA.x, fS * vA.y};
    return vX;
}

__device__ float inline dot(const float2 &vX, const float2 &vY)
{
    return vX.x * vY.x + vX.y * vY.y;
}

__device__ float inline sumsq(const float2 &vX)
{
    return dot(vX, vX);
}

__device__ float inline length(const float2 &vX)
{
    return sqrt(sumsq(vX));
}

///////////////////////////////////////////////////////////////////////////////////
// float3
__device__ float3 inline make_float3(const float *pData)
{
    float3 vX = {pData[0], pData[1], pData[2]};
    return vX;
}

__device__ float3 inline make_float3(const float *pData, const int iIdx)
{
    float3 vX = {pData[iIdx], pData[iIdx + 1], pData[iIdx + 2]};
    return vX;
}

__device__ void inline assign_float3(float *pData, const int iIdx, const float3 &vX)
{
    pData[iIdx + 0] = vX.x;
    pData[iIdx + 1] = vX.y;
    pData[iIdx + 2] = vX.z;
}

__device__ float3 inline abs(const float3 &vX)
{
    float3 vY = {abs(vX.x), abs(vX.y), abs(vX.z)};
    return vY;
}

__device__ float3 inline max(const float3 &vX, const float3 &vY)
{
    float3 vZ = {max(vX.x, vY.x), max(vX.y, vY.y), max(vX.z, vY.z)};
    return vZ;
}

__device__ float inline dot(const float3 &vX, const float3 &vY)
{
    return vX.x * vY.x + vX.y * vY.y + vX.z * vY.z;
}

__device__ float inline sumsq(const float3 &vX)
{
    return dot(vX, vX);
}

__device__ float inline length(const float3 &vX)
{
    return sqrt(sumsq(vX));
}

__device__ float3 inline operator-(const float3 &vA, const float3 &vB)
{
    float3 vX = {vA.x - vB.x, vA.y - vB.y, vA.z - vB.z};
    return vX;
}

__device__ float3 inline operator+(const float3 &vA, const float3 &vB)
{
    float3 vX = {vA.x + vB.x, vA.y + vB.y, vA.z + vB.z};
    return vX;
}

__device__ float3 inline operator*(const float &fS, const float3 &vA)
{
    float3 vX = {fS * vA.x, fS * vA.y, fS * vA.z};
    return vX;
}

__device__ float3 inline operator*(const float3 &vA, const float &fS)
{
    float3 vX = {fS * vA.x, fS * vA.y, fS * vA.z};
    return vX;
}

__device__ float3 inline operator/(const float3 &vA, const float &fS)
{
    float3 vX = {vA.x / fS, vA.y / fS, vA.z / fS};
    return vX;
}

__device__ float3 inline operator/(const float3 &vA, const float3 &vB)
{
    float3 vX = {vA.x / vB.x, vA.y / vB.y, vA.z / vB.z};
    return vX;
}

__device__ bool inline is_zero(const float &fX)
{
    return abs(fX) <= ZERO;
}

__device__ bool inline is_zero(const float3 &vX)
{
    return is_zero(sumsq(vX));
}

__device__ bool inline is_negative(const float &fX)
{
    return fX < -ZERO;
}

__device__ bool inline is_positive(const float &fX)
{
    return fX > ZERO;
}

////////////////////////////////////////////////////////////////////////////////////////////////
// This is an inefficient (from a CUDA point of view) evaluation of optical flow from object ids and
// rendered local coordinates of the objects. However, it is still much faster than
// programming this directly in python.
template <int t_iStartX, int t_iStartY, int t_iRangeX, int t_iRangeY,
          int t_iSizeX, int t_iSizeY,
          int t_iSearchRadiusX, int t_iSearchRadiusY,
          int t_iPosChanCnt, int t_iPosRowStride,
          int t_iIdxChanCnt, int t_iIdxRowStride,
          int t_iSubPixChanCnt, int t_iSubPixRowStride>
__global__ void EvalFlow(const float *aPos1, const float *aPos2, const int *aObjIdx1, const int *aObjIdx2, int *piIdxMapXY, float *pfSubPixIdx)
{
    const float fNaN = nanf("");

    const int iTrgX = blockDim.x * blockIdx.x + threadIdx.x;
    const int iX = min(iTrgX, t_iRangeX - 1);

    const int iTrgY = blockIdx.y;
    const int iY = min(iTrgY, t_iRangeY - 1);

    const int iTrgIdxX1 = iX + t_iStartX;
    const int iIdxX1 = min(max(iTrgIdxX1, 0), t_iSizeX - 1);

    const int iTrgIdxY1 = iY + t_iStartY;
    const int iIdxY1 = min(max(iTrgIdxY1, 0), t_iSizeY - 1);

    const int iPosIdx1 = t_iPosRowStride * iIdxY1 + t_iPosChanCnt * iIdxX1;
    const int iPosObjIdx1 = t_iSizeX * iIdxY1 + iIdxX1;
    const int iObjIdx1 = aObjIdx1[iPosObjIdx1];

    const bool bValidSrcIdx = (iObjIdx1 >= 0 && iTrgIdxX1 == iIdxX1 && iTrgIdxY1 == iIdxY1 && iTrgX == iX && iTrgY == iY);

    const float3 vPos1 = make_float3(aPos1, iPosIdx1);
    float3 vMaxDiff = make_float3(1e-3, 1e-3, 1e-3);

    // find the maximal gradient for each position dimension separately
    for (int iOffY = -1; iOffY <= 1; iOffY++)
    {
        const int iIterIdxY2 = iIdxY1 + iOffY;
        const int iIdxY2 = min(max(iIterIdxY2, 0), t_iSizeY - 1);
        const int iPosIdxY2 = t_iPosRowStride * iIdxY2;
        const bool bValidY = (bValidSrcIdx && iIterIdxY2 == iIdxY2);

        for (int iOffX = -1; iOffX <= 1; iOffX++)
        {
            const int iIterIdxX2 = iIdxX1 + iOffX;
            const int iIdxX2 = min(max(iIterIdxX2, 0), t_iSizeX - 1);

            const int iPosIdx2 = iPosIdxY2 + t_iPosChanCnt * iIdxX2;
            const int iPosObjIdx2 = t_iSizeX * iIdxY2 + iIdxX2;

            const bool bValid = (bValidY && iIterIdxX2 == iIdxX2 && iObjIdx1 == aObjIdx1[iPosObjIdx2]);

            const float3 vPos2 = make_float3(aPos1, iPosIdx2);
            const float3 vDiff = abs(vPos1 - vPos2);

            if (bValid)
            {
                vMaxDiff = max(vDiff, vMaxDiff);
            }
        }
    }

    vMaxDiff = vMaxDiff / max(vMaxDiff.x, max(vMaxDiff.y, vMaxDiff.z));

    float fMinValue = 1e38;
    int iMinPosIdx2 = -1;
    // int iMinObjIdx2 = -1;

    for (int iOffY = -t_iSearchRadiusY; iOffY <= t_iSearchRadiusY; iOffY++)
    {
        const int iIterIdxY2 = iIdxY1 + iOffY;
        const int iIdxY2 = min(max(iIterIdxY2, 0), t_iSizeY - 1);
        const int iPosIdxY2 = t_iPosRowStride * iIdxY2;
        bool bValidY = (bValidSrcIdx && iIterIdxY2 == iIdxY2);

        for (int iOffX = -t_iSearchRadiusX; iOffX <= t_iSearchRadiusX; iOffX++)
        {
            const int iIterIdxX2 = iIdxX1 + iOffX;
            const int iIdxX2 = min(max(iIterIdxX2, 0), t_iSizeX - 1);

            const int iPosIdx2 = iPosIdxY2 + t_iPosChanCnt * iIdxX2;
            const int iPosObjIdx2 = t_iSizeX * iIdxY2 + iIdxX2;

            bool bValid = (bValidY && iIterIdxX2 == iIdxX2 && iObjIdx1 == aObjIdx2[iPosObjIdx2]);

            float fValue = 0.0;
            float3 vPos2 = make_float3(aPos2, iPosIdx2);
            fValue = sqrt(sumsq((vPos1 - vPos2) / vMaxDiff));

            if (bValid && fValue < fMinValue)
            {
                fMinValue = fValue;
                iMinPosIdx2 = iPosIdx2;
                // iMinObjIdx2 = aObjIdx2[iPosObjIdx2];
            }
        }
    }

    const int iMapX = (iMinPosIdx2 % t_iPosRowStride) / t_iPosChanCnt;
    const int iMapY = iMinPosIdx2 / t_iPosRowStride;
    const int iMapIdxPos = iY * t_iIdxRowStride + iX * t_iIdxChanCnt;

    // Check whether all indices have been set at all
    bool bValid = bValidSrcIdx && iMinPosIdx2 >= 0;

    if (bValid)
    {
        const int iSubPixIdxPos = iY * t_iSubPixRowStride + iX * t_iSubPixChanCnt;
        const float3 vPosCtr = make_float3(aPos2, iMinPosIdx2);
        const float3 vVecTrg = vPos1 - vPosCtr;
        // float fIdxH = 0.0, fIdxV = 0.0;
        float2 vSubPix = make_float2(0.0, 0.0);
        bool bSubPixValid = true;

        if (!is_zero(vVecTrg))
        {
            bool pbValid[4];
            float3 pvPos[4];
            const int piH[4] = {1, 0, -1, 0};
            const int piV[4] = {0, -1, 0, 1};

            bSubPixValid = false;

            for (int i = 0; i < 4; i++)
            {
                const int iTrgMapX = iMapX + piH[i];
                const int iMapX = max(0, min(iTrgMapX, t_iSizeX - 1));

                const int iTrgMapY = iMapY + piV[i];
                const int iMapY = max(0, min(iTrgMapY, t_iSizeY - 1));

                const int iPosObjIdx = t_iSizeX * iMapY + iMapX;
                const int iPosIdx = t_iPosRowStride * iMapY + t_iPosChanCnt * iMapX;

                pbValid[i] = (iObjIdx1 == aObjIdx2[iPosObjIdx] && iMapX == iTrgMapX && iMapY == iTrgMapY);
                pvPos[i] = make_float3(aPos2, iPosIdx);
            }

            for (int i = 0; i < 4; i++)
            {
                const int iNext = (i + 1) % 4;
                const float3 vP1 = pvPos[i];
                const float3 vP2 = pvPos[iNext];

                if (pbValid[i] && pbValid[iNext])
                {
                    // Evaluate Barycentric Coordinates
                    const float3 vB1 = vP1 - vPosCtr;
                    const float3 vB2 = vP2 - vPosCtr;
                    const float2 vC = make_float2(dot(vVecTrg, vB1), dot(vVecTrg, vB2));
                    const float fB11 = dot(vB1, vB1);
                    const float fB12 = dot(vB1, vB2);
                    const float fB22 = dot(vB2, vB2);
                    const float fDetB = fB11 * fB22 - fB12 * fB12;
                    if (!is_zero(fDetB))
                    {
                        const float fL1 = (fB22 * vC.x - fB12 * vC.y) / fDetB;
                        const float fL2 = (fB11 * vC.y - fB12 * vC.x) / fDetB;
                        // It seems we do not always find the best central pixel.
                        // Therefore, fL1+fL2 can be greater than 1. Results look good nonetheless.
                        if ((fL1 >= 0.0 && fL2 >= 0.0) && (4.0 - fL1 - fL2) >= -1e-4)
                        {
                            vSubPix = fL1 * make_float2(float(piH[i]), float(piV[i])) + fL2 * make_float2(float(piH[iNext]), float(piV[iNext]));
                            // vSubPix = make_float2(fL1, fL2);
                            bSubPixValid = true;
                            break;
                        }
                    }
                }
            }
        }

        piIdxMapXY[iMapIdxPos + 0] = iObjIdx1;
        piIdxMapXY[iMapIdxPos + 1] = iIdxX1;
        piIdxMapXY[iMapIdxPos + 2] = iIdxY1;

        if (bSubPixValid)
        {
            const float fMapX = float(iMapX) + vSubPix.x;
            const float fMapY = float(iMapY) + vSubPix.y;

            pfSubPixIdx[iSubPixIdxPos + 0] = fMapX;
            pfSubPixIdx[iSubPixIdxPos + 1] = fMapY;
            pfSubPixIdx[iSubPixIdxPos + 2] = fMapX - float(iIdxX1);
            pfSubPixIdx[iSubPixIdxPos + 3] = fMapY - float(iIdxY1);
            // pfSubPixIdx[iSubPixIdxPos + 2] = vSubPix.x;
            // pfSubPixIdx[iSubPixIdxPos + 3] = vSubPix.y;

            piIdxMapXY[iMapIdxPos + 3] = iMapX;
            piIdxMapXY[iMapIdxPos + 4] = iMapY;
        }
        else
        {
            pfSubPixIdx[iSubPixIdxPos + 0] = fNaN;
            pfSubPixIdx[iSubPixIdxPos + 1] = fNaN;
            pfSubPixIdx[iSubPixIdxPos + 2] = fNaN;
            pfSubPixIdx[iSubPixIdxPos + 3] = fNaN;

            piIdxMapXY[iMapIdxPos + 3] = -1;
            piIdxMapXY[iMapIdxPos + 4] = -1;
        }
        // assign_float3(pfSubPixIdx, iIdxPos + 2, vPosCtr);
        // assign_float3(pfSubPixIdx, iIdxPos + 5, vPosR);

    } // if (bValid)
    else if (bValidSrcIdx)
    {
        piIdxMapXY[iMapIdxPos + 0] = -1;
        piIdxMapXY[iMapIdxPos + 1] = -1;
        piIdxMapXY[iMapIdxPos + 2] = -1;
        piIdxMapXY[iMapIdxPos + 3] = -1;
        piIdxMapXY[iMapIdxPos + 4] = -1;
        piIdxMapXY[iMapIdxPos + 5] = -1;
    }
}
